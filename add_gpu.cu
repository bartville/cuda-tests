
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


__global__
void add(int n, float* x, float* y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;  
  for(int i = index; i < n; i+=stride)
    y[i] += x[i];
}

int main(void) {

  int N = 1 << 20; //1 Milion elems
  float *x;
  float *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float)); 

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }  

  int blockSize = 256;
  int numBlocks = (N + blockSize-1) / blockSize;
  add<<< numBlocks, blockSize >>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}